#include "hip/hip_runtime.h"
#ifndef _CUDA_KERNELS_H_
#define _CUDA_KERNELS_H_

#ifdef __DEVICE_EMULATION__
#include <stdio.h>
#endif

// declare the texture reference for Matrix multiplication
texture<float, 2> transformTexture;

// convert a half-float to a single precision float
__device__ float halfToFloat (unsigned short halfIn)
{
	int s = (halfIn >> 15) & 0x00000001;
    int e = (halfIn >> 10) & 0x0000001f;
    int m =  halfIn        & 0x000003ff;

	int val = 0;
	int done = 0;

    if (e == 0)
    {
		if (m == 0)
		{
	    	//
	    	// Plus or minus zero
	    	//
			val = s << 31;
			done = 1;
		}
		else
		{
	    	//
	    	// Denormalized number -- renormalize it
	    	//

	    	while (!(m & 0x00000400))
	    	{
				m <<= 1;
				e -=  1;
	    	}

	    	e += 1;
	    	m &= ~0x00000400;
		}
    }
    else if (e == 31)
    {
		if (m == 0)
		{
	    	//
	    	// Positive or negative infinity
	    	//

	    	val = (s << 31) | 0x7f800000;
	    	done = 1;
		}
		else
		{
	    	//
	    	// Nan -- preserve sign and significand bits
	    	//

	    	val = (s << 31) | 0x7f800000 | (m << 13);
	    	done = 1;
		}
    }

    //
    // Normalized number
    //
	if ( !done )
	{
		e = e + (127 - 15);
    	m = m << 13;

    	//
    	// Assemble s, e and m.
    	//
		val = (s << 31) | (e << 23) | m;
	}

	return __int_as_float (val);

}

// convert a single precision float to a half-float
__device__ unsigned short floatToHalf( float floatIn )
{
    //
    // Our floating point number, f, is represented by the bit
    // pattern in integer i.  Disassemble that bit pattern into
    // the sign, s, the exponent, e, and the significand, m.
    // Shift s into the position where it will go in in the
    // resulting half number.
    // Adjust e, accounting for the different exponent bias
    // of float and half (127 versus 15).
    //

	int floatBits = __float_as_int( floatIn );

    int s =  ((floatBits >> 16) & 0x00008000);
    int e = ((floatBits >> 23) & 0x000000ff) - (127 - 15);
    int m =   floatBits        & 0x007fffff;
    //
    // Now reassemble s, e and m into a half:
    //

    if (e <= 0)
    {
		if (e < -10)
		{
	    	//
	    	// E is less than -10.  The absolute value of f is
	    	// less than HALF_MIN (f may be a small normalized
	    	// float, a denormalized float or a zero).
	    	//
	    	// We convert f to a half zero.
	    	//

	    	return 0;
		}

		//
		// E is between -10 and 0.  F is a normalized float,
		// whose magnitude is less than HALF_NRM_MIN.
		//
		// We convert f to a denormalized half.
		//

		m = (m | 0x00800000) >> (1 - e);

		//
		// Round to nearest, round "0.5" up.
		//
		// Rounding may cause the significand to overflow and make
		// our number normalized.  Because of the way a half's bits
		// are laid out, we don't have to treat this case separately;
		// the code below will handle it correctly.
		//

		if (m &  0x00001000)
	    	m += 0x00002000;

		//
		// Assemble the half from s, e (zero) and m.
		//
		return s | (m >> 13);
    }
    else if (e == 0xff - (127 - 15))
    {
		if (m == 0)
		{
	    	//
	    	// F is an infinity; convert f to a half
	    	// infinity with the same sign as f.
	    	//
			return s | 0x7c00;
		}
		else
		{
	    	//
	    	// F is a NAN; produce a half NAN that preserves
	    	// the sign bit and the 10 leftmost bits of the
	    	// significand of f.
	    	//
			return s | 0x7c00 | (m >> 13);
		}
    }
    else
    {
		//
		// E is greater than zero.  F is a normalized float.
		// We try to convert f to a normalized half.
		//

		//
		// Round to nearest, round "0.5" up
		//

		if (m &  0x00001000)
		{
	    	m += 0x00002000;

	    	if (m & 0x00800000)
	    	{
				m =  0;		// overflow in significand,
				e += 1;		// adjust exponent
	    	}
		}

		//
		// Handle exponent overflow
		//

		if (e > 30)
		{
			return s | 0x7c00;	// if this returns, the half becomes an
	    	 // infinity with the same sign as f.
		}

		//
		// Assemble the half from s, e and m.
		//
		return s | (e << 10) | (m >> 13);
    }
}

__global__ void add_kernel (ushort4 *image_RGBA, int width, int height, float value)
{
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w
        const int idx = width * iy + ix;

        float4 pixelFloat;

        pixelFloat.x = halfToFloat((unsigned short)image_RGBA[idx].x) + value;
        pixelFloat.y = halfToFloat((unsigned short)image_RGBA[idx].y) + value;
        pixelFloat.z = halfToFloat((unsigned short)image_RGBA[idx].z) + value;

        image_RGBA[idx].x = floatToHalf(pixelFloat.x);
        image_RGBA[idx].y = floatToHalf(pixelFloat.y);
        image_RGBA[idx].z = floatToHalf(pixelFloat.z);
    }
	__syncthreads();
}

__global__ void multiply_kernel (ushort4 *image_RGBA, int width, int height, float value)
{
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w
        const int idx = width * iy + ix;

        float4 pixelFloat;

        pixelFloat.x = halfToFloat((unsigned short)image_RGBA[idx].x) * value;
        pixelFloat.y = halfToFloat((unsigned short)image_RGBA[idx].y) * value;
        pixelFloat.z = halfToFloat((unsigned short)image_RGBA[idx].z) * value;

        image_RGBA[idx].x = floatToHalf(pixelFloat.x);
        image_RGBA[idx].y = floatToHalf(pixelFloat.y);
        image_RGBA[idx].z = floatToHalf(pixelFloat.z);
    }
	__syncthreads();
}

/**
 * Apply gamma transform to each pixel value of an immage
 */
__global__ void gamma_kernel (ushort4 *image_RGBA, int width, int height, float value)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w
        const int idx = width * iy + ix;

        float4 pixelFloat;

        pixelFloat.x = powf(halfToFloat((unsigned short)image_RGBA[idx].x), value);
        pixelFloat.y = powf(halfToFloat((unsigned short)image_RGBA[idx].y), value);
        pixelFloat.z = powf(halfToFloat((unsigned short)image_RGBA[idx].z), value);

        image_RGBA[idx].x = floatToHalf(pixelFloat.x);
        image_RGBA[idx].y = floatToHalf(pixelFloat.y);
        image_RGBA[idx].z = floatToHalf(pixelFloat.z);
    }
}

/**
 * Invert each color channel value:  out = max - in + min  with max = 1 and min = 0
 */
__global__ void invert_kernel (ushort4 *image_RGBA, int width, int height)
{
    #define MAX_HALF 1.0f

    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w
        const int idx = width * iy + ix;

        float4 pixelFloat;

        pixelFloat.x = MAX_HALF - halfToFloat((unsigned short)image_RGBA[idx].x);
        pixelFloat.y = MAX_HALF - halfToFloat((unsigned short)image_RGBA[idx].y);
        pixelFloat.z = MAX_HALF - halfToFloat((unsigned short)image_RGBA[idx].z);

        image_RGBA[idx].x = floatToHalf(pixelFloat.x);
        image_RGBA[idx].y = floatToHalf(pixelFloat.y);
        image_RGBA[idx].z = floatToHalf(pixelFloat.z);
    }
}

/**
 * Create an alpha channel from the color difference
 */
__global__ void matte_color_diff_kernel (ushort4 *image_RGBA, int width, int height, float threshold)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w
        const int idx = width * iy + ix;

        float4 pixelFloat;

        pixelFloat.x = halfToFloat((unsigned short)image_RGBA[idx].x);
        pixelFloat.y = halfToFloat((unsigned short)image_RGBA[idx].y);
        pixelFloat.z = fminf(pixelFloat.y, halfToFloat((unsigned short)image_RGBA[idx].z));

        if ( pixelFloat.z > threshold )
        {
            pixelFloat.w = 1 - pixelFloat.z + fmaxf(pixelFloat.x, pixelFloat.y);
        }
        else
        {
            pixelFloat.w = 1.0f;
        }

        // red and green channels are unchanged
        image_RGBA[idx].z = floatToHalf(pixelFloat.z);
        image_RGBA[idx].w = floatToHalf(pixelFloat.w);

    }
}

/**
 * Invert the alpha channel:  out = max - in + min  with max = 1 and min = 0
 */
__global__ void matte_invert_kernel (ushort4 *image_RGBA, int width, int height)
{
    #define MAX_HALF 1.0f

    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w
        const int idx = width * iy + ix;

        float newAlpha;

        newAlpha = MAX_HALF - halfToFloat((unsigned short)image_RGBA[idx].w);

        image_RGBA[idx].w = floatToHalf(newAlpha);
    }
}

__global__ void color_monochrome_kernel ( ushort4 *image_RGBA, int width, int height, float redWeight, float greenWeight, float blueWeight)
{
	const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w
        const int idx = width * iy + ix;

        float monoValue;

        monoValue = halfToFloat(image_RGBA[idx].x) * redWeight
        		  + halfToFloat(image_RGBA[idx].y) * greenWeight
        		  + halfToFloat(image_RGBA[idx].z) * blueWeight;

        image_RGBA[idx].x = floatToHalf(monoValue);
        image_RGBA[idx].y = floatToHalf(monoValue);
        image_RGBA[idx].z = floatToHalf(monoValue);
    }
	__syncthreads();
}

__global__ void threshold_kernel ( ushort4 *image_RGBA, int width, int height, float redThreshold, float greenThreshold, float blueThreshold, float alphaThreshold)
{
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if(ix < width && iy < height)
    {
        // the first, second, third, and fourth fields can be accessed using x, y, z, and w
        const int idx = width * iy + ix;

        float4 pixelFloat;

        pixelFloat.x = fmaxf(halfToFloat((unsigned short)image_RGBA[idx].x), redThreshold);
        pixelFloat.y = fmaxf(halfToFloat((unsigned short)image_RGBA[idx].y), blueThreshold);
        pixelFloat.z = fmaxf(halfToFloat((unsigned short)image_RGBA[idx].z), greenThreshold);
        pixelFloat.w = fmaxf(halfToFloat((unsigned short)image_RGBA[idx].w), alphaThreshold);

        image_RGBA[idx].x = floatToHalf(pixelFloat.x);
        image_RGBA[idx].y = floatToHalf(pixelFloat.y);
        image_RGBA[idx].z = floatToHalf(pixelFloat.z);
        image_RGBA[idx].w = floatToHalf(pixelFloat.w);
    }
    __syncthreads();
}

__global__ void linear_transform_kernel ( float4 *points, int num_points )
{
	const int idx = blockDim.x * blockIdx.x + threadIdx.x;
	__shared__ float T[4][4];

	__shared__ float4 ThreadPoints[64];

	/*
	__syncthreads();

	if ( threadIdx.x < 1 )
	{
		T[0][0] = tex2D(transformTexture, 0, 0);
		T[1][0] = tex2D(transformTexture, 0, 1);
		T[2][0] = tex2D(transformTexture, 0, 2);
		T[3][0] = tex2D(transformTexture, 0, 3);
		T[0][1] = tex2D(transformTexture, 1, 0);
		T[1][1] = tex2D(transformTexture, 1, 1);
		T[2][1] = tex2D(transformTexture, 1, 2);
		T[3][1] = tex2D(transformTexture, 1, 3);
		T[0][2] = tex2D(transformTexture, 2, 0);
		T[1][2] = tex2D(transformTexture, 2, 1);
		T[2][2] = tex2D(transformTexture, 2, 2);
		T[3][2] = tex2D(transformTexture, 2, 3);
		T[0][3] = tex2D(transformTexture, 3, 0);
		T[1][3] = tex2D(transformTexture, 3, 1);
		T[2][3] = tex2D(transformTexture, 3, 2);
		T[3][3] = tex2D(transformTexture, 3, 3);
	}
	*/

    if ( threadIdx.x < 16 )
	{
		//const int row = idx >> 2;
		//const int col = idx & 0x3;
		T[idx >> 2][idx & 0x3] = tex2D(transformTexture, idx & 0x3, idx >> 2);
	}
	if ( blockDim.x < 16 )
	{
		for ( int i = blockDim.x ; i < 16 ; i++ )
		{
			T[i >> 2][i & 0x3] = tex2D(transformTexture, i & 0x3, i >> 2);
		}
	}
	__syncthreads();

	if ( idx < num_points )
	{
		float4 vt;
		ThreadPoints[threadIdx.x] = points[idx];
		vt.w = T[3][0]*ThreadPoints[threadIdx.x].x + T[3][1]*ThreadPoints[threadIdx.x].y + T[3][2]*ThreadPoints[threadIdx.x].z + T[3][3];

		vt.x = (T[0][0]*ThreadPoints[threadIdx.x].x + T[0][1]*ThreadPoints[threadIdx.x].y + T[0][2]*ThreadPoints[threadIdx.x].z + T[0][3])/vt.w;
		vt.y = (T[1][0]*ThreadPoints[threadIdx.x].x + T[1][1]*ThreadPoints[threadIdx.x].y + T[1][2]*ThreadPoints[threadIdx.x].z + T[1][3])/vt.w;
		vt.z = (T[2][0]*ThreadPoints[threadIdx.x].x + T[2][1]*ThreadPoints[threadIdx.x].y + T[2][2]*ThreadPoints[threadIdx.x].z + T[2][3])/vt.w;

		points[idx].x = ThreadPoints[threadIdx.x].x*(1 - ThreadPoints[threadIdx.x].w) + ThreadPoints[threadIdx.x].w*vt.x;
		points[idx].y = ThreadPoints[threadIdx.x].y*(1 - ThreadPoints[threadIdx.x].w) + ThreadPoints[threadIdx.x].w*vt.y;
		points[idx].z = ThreadPoints[threadIdx.x].z*(1 - ThreadPoints[threadIdx.x].w) + ThreadPoints[threadIdx.x].w*vt.z;
	}

	__syncthreads();

}

/**
 * Kernel for calculating the coordinates of the new points along the specified edges.
 *
 */
__global__ void subdivide_edges_split_point_kernel ( const unsigned int* first_midpoint,
																											const unsigned char* has_midpoint,
                                                     float4* points_and_selection,
                                                     unsigned int num_points,
                                                     unsigned int* edge_point_indices,
                                                     unsigned int* clockwise_edge_indices,
                                                     const float* edge_selection,
                                                     const unsigned int* companions,
                                                     const unsigned char* boundary_edges,
                                                     const unsigned int* edge_faces,
                                                     int num_split_points,
                                                     const int num_edges)
{
    unsigned int edge_index = (blockIdx.x * blockDim.x) + threadIdx.x;
    int split_index = (blockIdx.y * blockDim.y) + threadIdx.y;

    const unsigned char boundary = boundary_edges[edge_index];
    const unsigned int companion = companions[edge_index];
    if ( edge_index < num_edges  && has_midpoint[edge_index] && (boundary || (!boundary && (edge_faces[companion] > edge_faces[edge_index] || !edge_selection[companion]))))
    {
        unsigned int p_index = edge_point_indices[edge_index];
        unsigned int new_point_index = first_midpoint[edge_index] + split_index;

        #ifdef __DEVICE_EMULATION__
	/*
            printf("Split Index: %d\n", split_index);
            printf("Edge Index: %d\n", edge_index);
            printf("Clockwise Edge Index: %d\n", clockwise_edge_indices[edge_index]);
            printf("Point Index: %d\n", edge_point_indices[edge_index]);
            printf("Clockwise Point Index: %d\n", edge_point_indices[clockwise_edge_indices[edge_index]]);
            printf("New Point Index: %d\n", new_point_index);
	*/
        #endif


        float4 p0 = points_and_selection[p_index];
        float4 p1 = points_and_selection[edge_point_indices[clockwise_edge_indices[edge_index]]];

        #ifdef __DEVICE_EMULATION__
	/*
            printf("P_0:%d: (%f, %f, %f)\n", p_index, p0.x, p0.y, p0.z);
            printf("P_1:%d: (%f, %f, %f)\n", edge_point_indices[clockwise_edge_indices[edge_index]], p1.x, p1.y, p1.z);
	*/
        #endif

        p1.x = (p1.x - p0.x) / (num_split_points + 1);
        p1.y = (p1.y - p0.y) / (num_split_points + 1);
        p1.z = (p1.z - p0.z) / (num_split_points + 1);

        #ifdef __DEVICE_EMULATION__
	/*
            printf("P_delta:(%f, %f, %f)\n", p1.x, p1.y, p1.z);
	*/
        #endif

        points_and_selection[new_point_index].x = p0.x + (split_index + 1)*p1.x;
        points_and_selection[new_point_index].y = p0.y + (split_index + 1)*p1.y;
        points_and_selection[new_point_index].z = p0.z + (split_index + 1)*p1.z;
        points_and_selection[new_point_index].w = 1;

    }

}

__global__ void subdivide_edges_update_edge_indices_kernel ( unsigned int* output_edge_point_indices,
                                                        unsigned int* output_clockwise_edge_point_indices,
                                                        unsigned int* input_edge_point_indices,
                                                        unsigned int* input_clockwise_edge_point_indices,
                                                        unsigned int* edge_index_map,
                                                        int num_edge_maps)
{
    unsigned int edge_index_index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if ( edge_index_index < num_edge_maps )
    {
        unsigned int out_edge_index = edge_index_map[edge_index_index];

        #ifdef __DEVICE_EMULATION__
	/*
            printf("Edge: %d : Mapped Edge : %d \n", edge_index_index, out_edge_index);
	*/
        #endif

        output_edge_point_indices[out_edge_index] = input_edge_point_indices[edge_index_index];
        output_clockwise_edge_point_indices[out_edge_index] = edge_index_map[input_clockwise_edge_point_indices[edge_index_index]];

        #ifdef __DEVICE_EMULATION__
	/*
            printf("Input Edge Point: %d : Input CW Edge Point : %d \n", input_edge_point_indices[edge_index_index], input_clockwise_edge_point_indices[edge_index_index]);
	*/
        #endif
    }
}

__global__ void subdivide_edges_update_loop_first_edges_kernel (
                                                        unsigned int* pdev_ouput_loop_first_edges,
                                                        int num_loops,
                                                        unsigned int* edge_index_map
                                                        )
{
    unsigned int loop_index = (blockIdx.x * blockDim.x) + threadIdx.x;

    if ( loop_index < num_loops )
    {
        pdev_ouput_loop_first_edges[loop_index] = edge_index_map[pdev_ouput_loop_first_edges[loop_index]];
    }
}

__global__ void subdivide_edges_split_edges_kernel (unsigned int* output_edge_point_indices,
                                                    unsigned int* output_clockwise_edge_point_indices,
                                                    unsigned int* input_clockwise_edge_point_indices,
                                                    const float* edge_selection,
                                                    unsigned int* edge_index_map,
                                                    unsigned int* pdev_first_midpoint,
                                                    unsigned char* has_midpoint,
                                                    int num_split_points,
                                                    unsigned int* pdev_companions,
                                                    unsigned char* pdev_boundary_edges,
                                                    const unsigned int num_edges,
                                                    const unsigned int* edge_faces
                                                    )

{
    unsigned int edge = (blockIdx.x * blockDim.x) + threadIdx.x;
    int split_index = (blockIdx.y * blockDim.y) + threadIdx.y;
    
    if ( edge < num_edges  && has_midpoint[edge])
    {
        unsigned int old_clockwise = input_clockwise_edge_point_indices[edge];
        unsigned int new_edge = edge_index_map[edge] + 1 + split_index;
        
        const unsigned int companion = pdev_companions[edge];
        const unsigned char boundary = pdev_boundary_edges[edge];

        if(edge_selection[edge] && (boundary || (!boundary && (edge_faces[edge] < edge_faces[companion] || !edge_selection[companion]))))
        {
        	output_edge_point_indices[new_edge] = pdev_first_midpoint[edge] + split_index;
        }
        else
        {
        	output_edge_point_indices[new_edge] = pdev_first_midpoint[edge] - split_index + num_split_points - 1;
        }
        output_clockwise_edge_point_indices[new_edge - 1] = new_edge;

        if ( split_index == 0 )
        {
            output_clockwise_edge_point_indices[new_edge + num_split_points - 1] = edge_index_map[old_clockwise];
        }

    }
}

__global__ void convert_uint_64_to_32_kernel ( uint2* p_uint_64, unsigned int* p_uint_32, int num_ints )
{
    const int int_index = (blockDim.x * blockIdx.x) + threadIdx.x;

    if ( int_index < num_ints )
    {
        // set the 32bit unsigned int to the lower 32bits of the 64bit unsigned int
        p_uint_32[int_index] = p_uint_64[int_index].x;
    }
}

__global__ void convert_uint_32_to_64_kernel ( uint2* p_uint_64, unsigned int* p_uint_32, int num_ints )
{
    const int int_index = (blockDim.x * blockIdx.x) + threadIdx.x;

    if ( int_index < num_ints )
    {
        // zero the upper 32bits and equate the lower 32
        p_uint_64[int_index].y = 0;
        p_uint_64[int_index].x = p_uint_32[int_index];
    }
}


__global__ void set_selection_value_kernel ( float4* points_and_selection, float selection_value, int num_points )
{
    const int index = (blockDim.x * blockIdx.x) + threadIdx.x;
    if ( index < num_points )
    {
        points_and_selection[index].w = selection_value;
    }
}

/**
 * Kernel for companion computation
 */
__global__ void find_companion_kernel(unsigned char* boundary_edges,
									  unsigned int* adjacent_edge_indices,
									  const int num_edges,
									  const unsigned int* edge_point_indices,
									  const unsigned int* clockwise_edges_point_indices,
									  const unsigned int* first_edges,
									  const unsigned int* valences,
									  const unsigned int* point_edges)
{
	const int edge_index = (blockDim.x * blockIdx.x) + threadIdx.x;

	if ( edge_index < num_edges )
	{
		const unsigned int vertex1 = edge_point_indices[edge_index];
		const unsigned int vertex2 = edge_point_indices[clockwise_edges_point_indices[edge_index]];

		const unsigned int first_index = first_edges[vertex2];
		const unsigned int last_index = first_index + valences[vertex2];
		for ( unsigned int i = first_index; i != last_index; ++i )
		{
			const unsigned int companion = point_edges[i];
			if ( edge_point_indices[clockwise_edges_point_indices[companion]] == vertex1 )
			{
				boundary_edges[edge_index] = 0;
				adjacent_edge_indices[edge_index] = companion;
				break;
			}
		}
	}

}


/**
 * Kernel for looking up the valence of edges points
 */
__global__ void create_vertex_valence_lookup_kernel ( unsigned int* valence, const unsigned int* edge_point_indices, int* num_valence, int num_edges )
{
	int tmp_num_valence = 0;
	unsigned int point_index = 0;

	for ( unsigned int edge_index = 0; edge_index < num_edges ; edge_index++ )
	{
		point_index = edge_point_indices[edge_index];
		valence[point_index] += 1;
		tmp_num_valence = max(tmp_num_valence, (point_index+1));
	}

	*num_valence = tmp_num_valence;
}

/**
 * Kernel to calculate first edges in edge_adjacency lookup
 */
__global__ void calculate_first_edges_kernel ( unsigned int* first_edges, const unsigned int* valences, int num_points )
{
	unsigned int count = 0;

	for ( unsigned int point = 0; point < num_points; ++point)
	{
		first_edges[point] = count;
		count += valences[point];
	}

}
__global__ void calculate_first_edges_block_kernel ( unsigned int* first_edges, const unsigned int* valences, int num_edges )
{
	// use shared memory to store both the result per block as well as the initial block values
	__shared__ extern uint2 shared_first[];

	unsigned int edge_index = ( blockDim.x * blockIdx.x) + threadIdx.x;

	if ( edge_index < num_edges )
	{
		shared_first[threadIdx.x].x = valences[edge_index];
		shared_first[0].y = 0;

		if ( edge_index > 0 )
		{
			shared_first[threadIdx.x].y = valences[edge_index-1];
		}
	}

	__syncthreads();

	if ( edge_index < num_edges )
	{
		for ( int i = 1; i < blockDim.x ; i++ )
		{
			if ( threadIdx.x >= i )
			{
				shared_first[threadIdx.x].x += shared_first[threadIdx.x - i].y;
			}
		}
	}

	__syncthreads();
	if ( edge_index < num_edges )
	{
		first_edges[edge_index] = shared_first[threadIdx.x].x;
	}
	__syncthreads();

}

__global__ void calculate_first_edges_update_kernel ( unsigned int* first_edges, int num_edges )
{

	__shared__ extern unsigned int shared_first_edge[];
	__shared__ unsigned int update_val;

	unsigned int edge_index = ( blockDim.x * blockIdx.x) + threadIdx.x;

#ifdef __DEVICE_EMULATION__
/*
	printf("calculate_first_edges_update_kernel: %u of %d blah\n", edge_index, num_edges);
*/
#endif

	if ( edge_index < num_edges )
	{
		shared_first_edge[threadIdx.x] = first_edges[edge_index];
	}

	if ( threadIdx.x == 0 )
	{
		update_val = 0;
		for ( int k = 1 ; k <= ( blockIdx.x ) ; k++ )
		{
			update_val += first_edges[k*blockDim.x -1];
		}
	}
	__syncthreads();


	if ( edge_index < num_edges )
	{
		shared_first_edge[threadIdx.x] += update_val;
	}

	__syncthreads();

	if ( edge_index < num_edges )
	{
		first_edges[edge_index] = shared_first_edge[threadIdx.x];
	}

}

/**
 * Kernel for calculating point edges
 */
__global__ void calculate_point_edges_kernel ( unsigned int* point_edges, unsigned int* found_edges, const unsigned int* edge_point_indices, const unsigned int* first_edges, int num_edges)
{
	unsigned int point_index = 0;
	for ( unsigned int edge_index = 0 ; edge_index < num_edges ; ++edge_index )
	{
		point_index = edge_point_indices[edge_index];
		point_edges[first_edges[point_index] + found_edges[point_index]] = edge_index;
		++found_edges[point_index];
	}
}

/**
 * Kernel to get the first loop index as well as the loop count for a given face
 */
__global__ void get_loop_index_and_count_kernel (
											unsigned int* index_and_counts,
											const unsigned int* face_first_loops,
											const unsigned int* face_loop_counts,
											int face_index )
{
	index_and_counts[0] = face_first_loops[face_index];
	index_and_counts[1] = face_loop_counts[face_index];
}


/**
 * Count the total number of edges each face will have after applying the edge subdivision, and the number of new points added per face
 */

__global__ void subdivide_edges_count_components_per_face(
													const unsigned int* face_first_loops,
													const int num_faces,
													const unsigned int* face_loop_counts,
													const unsigned int* loop_first_edges,
													const unsigned int* clockwise_edges,
													const float* edge_selection,
													const unsigned int* companions,
													const unsigned char* boundary_edges,
													const int split_point_count,
													const unsigned int* edge_faces,
													unsigned int* face_edge_counts,
													unsigned int* face_split_point_counts
		)
{
	unsigned int face_index = (blockIdx.x * blockDim.x) + threadIdx.x;	
	if(face_index >= num_faces)
		return;
	
	unsigned int edge_count = 0;
	unsigned int new_point_count = 0;
	
	unsigned int loop_begin = face_first_loops[face_index];
	unsigned int loop_end = loop_begin + face_loop_counts[face_index];
	for(unsigned int loop_index = loop_begin; loop_index != loop_end; ++loop_index)
	{
		unsigned int first_edge_index = loop_first_edges[loop_index];
		
		for( unsigned int edge_index = first_edge_index; 1 ; )
		{
			unsigned int companion = companions[edge_index]; 
			++edge_count;
	
			edge_count += split_point_count * (edge_selection[edge_index] || (!boundary_edges[edge_index] && edge_selection[companion]));
			unsigned int found_points = 0;
			if(edge_selection[edge_index] && (boundary_edges[edge_index] || edge_faces[edge_index] < edge_faces[companion] || !edge_selection[companion]))
				found_points = split_point_count;
			//const unsigned int found_points = split_point_count * (edge_selection[edge_index] && !(!boundary_edges[edge_index] && (edge_faces[edge_index] > edge_faces[companion] || !edge_selection[companion])));
			new_point_count += found_points;
			
#ifdef __DEVICE_EMULATION__
/*
	printf("adding %d points for edge %d\n", found_points, edge_index);
*/
#endif
						
			edge_index = clockwise_edges[edge_index];
			if (edge_index == first_edge_index)
				break;
		}
	}
	
	face_edge_counts[face_index] = edge_count;
	face_split_point_counts[face_index] = new_point_count;
}

/**
 *
 */
__global__ void subdivide_edges_edge_index_calculator_kernel (
											unsigned int* first_midpoint,
											unsigned char* has_midpoint,
											unsigned int* index_map,
											const unsigned int* face_first_loops,
											int num_faces,
											const unsigned int* face_loop_counts,
											const unsigned int* loop_first_edges,
											const unsigned int* clockwise_edges,
											const float* edge_selection,
											const unsigned int* companions,
											const unsigned char* boundary_edges,
											const unsigned int* edge_faces,
											const unsigned int* face_edge_counts,
											const unsigned int* face_split_point_counts,
											int split_point_count,
											int first_new_point_index
											 )
{

	unsigned int face_index = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if(face_index >= num_faces)
		return;
		
	// initialize the edge count to the cumulative edge count of the previous faces
	unsigned int edge_count = face_index == 0 ? 0 : face_edge_counts[face_index - 1];
	// initialise the new midpoint point index to the cumulative new point count of the previous faces
	unsigned int midpoint_index = face_index == 0 ? first_new_point_index : first_new_point_index + face_split_point_counts[face_index - 1];

	unsigned int loop_begin = face_first_loops[face_index];
	unsigned int loop_end = loop_begin + face_loop_counts[face_index];
	for(unsigned int loop_index = loop_begin; loop_index != loop_end; ++loop_index)
	{
		unsigned int first_edge_index = loop_first_edges[loop_index];
		for( unsigned int edge_index = first_edge_index; 1 ; )
		{
			index_map[edge_index] = edge_count;
			const unsigned int companion = companions[edge_index];
			const char boundary = boundary_edges[edge_index];
			
			// The midpoint was/is being/will be calculated in another kernel, but we still need to update the edge index counter for this face
			if(!boundary && edge_selection[companion] && (!edge_selection[edge_index] || edge_faces[edge_index] > edge_faces[companion]))
			{
				edge_count += split_point_count;
			}
	
			// Get the midpoint index, and set it for both ourselves and the companion edge if we are the kernel for the first face of the edge-companion pair
			if(edge_selection[edge_index] && (boundary || (!boundary && (edge_faces[edge_index] < edge_faces[companion] || !edge_selection[companion]))))
			{
				first_midpoint[edge_index] = midpoint_index;
				has_midpoint[edge_index] = 1;
				if(!boundary)
				{
					first_midpoint[companion] = midpoint_index;
					has_midpoint[companion] = 1;
				}
				midpoint_index += split_point_count;
				edge_count += split_point_count;
			}
	
			edge_index = clockwise_edges[edge_index];
			++edge_count;
			if (edge_index == first_edge_index)
				break;
		}
	}
}

/**
 * Execute a single step in the iterative calculation of a cumulative sum. The algorithm
 * has to as long as step_size < size
 * \param input The input array
 * \param output The result, which is partial as long as step_size < size
 * \param size The number of elements in input and output
 * \param step_size Equal to 2^(step_number), with step_number starting at 0
 */
__global__ void cumulative_sum_step(const unsigned int* input, unsigned int* output, const int size, const int step_size)
{
	const unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if(index >= size)
		return;
	
	char to_calculate = step_size <= index; // true if the number at index still needs calculating
	output[index] = input[index] + input[(index - step_size) * to_calculate] * to_calculate;
}

/**
 * Stores the face each edge belongs to
 */
__global__ void face_per_edge_kernel(
													const unsigned int* face_first_loops,
													const int num_faces,
													const unsigned int* face_loop_counts,
													const unsigned int* loop_first_edges,
													const unsigned int* clockwise_edges,
													unsigned int* edge_faces)
{
	unsigned int face_index = (blockIdx.x * blockDim.x) + threadIdx.x;	
	if(face_index >= num_faces)
		return;

	unsigned int loop_begin = face_first_loops[face_index];
	unsigned int loop_end = loop_begin + face_loop_counts[face_index];
	for(unsigned int loop_index = loop_begin; loop_index != loop_end; ++loop_index)
	{
		unsigned int first_edge_index = loop_first_edges[loop_index];
		
		for( unsigned int edge_index = first_edge_index; 1 ; )
		{
			edge_faces[edge_index] = face_index;
			
			edge_index = clockwise_edges[edge_index];
			if (edge_index == first_edge_index)
				break;
		}
	}
}

/**
 * Kernel to construct the mesh structure of the polyhedra
 */
__global__ void create_grid_structure_kernel (
												unsigned int* face_first_loops,
												unsigned int* face_loop_count,
												unsigned int* loop_first_edge,
												uint4* edge_point,
												uint4* clockwise_edge,
												unsigned int rows,
												unsigned int columns
												)
{
	unsigned int row_index = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	unsigned int col_index = ( blockIdx.y * blockDim.y ) + threadIdx.y;

	if ( ( row_index < rows ) && (col_index < columns) )
	{
		unsigned int face_number = row_index * columns + col_index;
		face_first_loops[face_number] = face_number;
		face_loop_count[face_number] = 1;
		loop_first_edge[face_number] = 4*face_number;

		edge_point[face_number].x = face_number + row_index;
		edge_point[face_number].y = face_number + row_index + 1;
		edge_point[face_number].z = face_number + row_index + columns + 2;
		edge_point[face_number].w = face_number + row_index + columns + 1;

		clockwise_edge[face_number].x = 4*face_number + 1;
		clockwise_edge[face_number].y = 4*face_number + 2;
		clockwise_edge[face_number].z = 4*face_number + 3;
		clockwise_edge[face_number].w = 4*face_number;

	}

}

/**
 * Kernel to calculate the point coordinates of a poly_grid
 */
__global__ void calculate_grid_points_kernel ( float4 *point_and_selection,
												float3 x,
												float3 y,
												unsigned int rows,
												unsigned int columns
												)
{
	unsigned int row_index = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	unsigned int col_index = ( blockIdx.y * blockDim.y ) + threadIdx.y;

	if ( ( row_index <= rows ) && (col_index <= columns) )
	{
		float row_percent = 0.5 - (float)row_index / (float)rows;
		float col_percent = (float)col_index/(float)columns - 0.5;

		unsigned int point_index = row_index*(columns+1) + col_index;

		point_and_selection[point_index].x = col_percent * x.x + row_percent * y.x;
		point_and_selection[point_index].y = col_percent * x.y + row_percent * y.y;
		point_and_selection[point_index].z = col_percent * x.z + row_percent * y.z;
		point_and_selection[point_index].w = 0;
	}

}

#endif // #ifndef _CUDA_KERNELS_H_

