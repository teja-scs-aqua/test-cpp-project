#include "hip/hip_runtime.h"
// K-3D
// Copyright (c) 1995-2008, Timothy M. Shead
//
// Contact: tshead@k-3d.com
//
// This program is free software; you can redistribute it and/or
// modify it under the terms of the GNU General Public
// License as published by the Free Software Foundation; either
// version 2 of the License, or (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
// General Public License for more details.
//
// You should have received a copy of the GNU General Public
// License along with this program; if not, write to the Free Software
// Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA

/** \file
    \author Evan Lezar (evanlezar@gmail.com)
*/

// cuda includes
#include <stdio.h>
#include <hip/hip_vector_types.h>
//include the kernels
#include "cuda_kernels.cu"

// define the externals
#include "cuda_entry_points.h"

#ifdef K3D_API_WIN32

/// Retrieves a timestamp in seconds using the Win32 high performance counters
inline double nanotime()
{
	LARGE_INTEGER timestamp;
	LARGE_INTEGER frequency;
	if ( !(QueryPerformanceCounter(&timestamp) && QueryPerformanceFrequency(&frequency)) )
	{
		return 0.0;
	}

	return static_cast<double>(timestamp.QuadPart) / static_cast<double>(frequency.QuadPart);
}

#else // K3D_API_WIN32

/// Retrieves a timestamp in seconds using gettimeofday() for portable timing
inline double nanotime()
{
	timeval tv;
	gettimeofday(&tv, 0);

	return tv.tv_sec + static_cast<double>(tv.tv_usec) / 1000000;
}

#endif // !K3D_API_WIN32


/**
 * Initialize the timing info structure
 */
void initTimingInfo(timingInfo_t* tInfo, int numberOfEntries)
{
	(*tInfo).numEntries = numberOfEntries;
	(*tInfo).timings = (double*)malloc ( numberOfEntries*sizeof(double) );
	(*tInfo).labels = (char**)malloc ( numberOfEntries*sizeof(char*) );
	for ( int i = 0 ; i < numberOfEntries ; i++ )
	{
		(*tInfo).labels[i] = (char*) malloc ( 33*sizeof(char) );
	}
}

/**
 * Set the label of a given timing_info entry
 */
inline void setTimingInfoLabel(timingInfo_t* tInfo, int index, char* label)
{
	sprintf((*tInfo).labels[index], "%s", label);
}

/**
 * Initialize the timing_info entry to the current time
 */
inline void startTimingInfoTimer (timingInfo_t* tInfo, int index)
{
	(*tInfo).timings[index] = nanotime();
}
/**
 * Set the timing value of a given timing_info entry to the elapsed time since it was started
 */
inline void measureTimingInfoTimer(timingInfo_t* tInfo, int index)
{
	(*tInfo).timings[index] = nanotime() - (*tInfo).timings[index];
}

/**
 * Integer division and rounding up
 */
int iDivUp(int a, int b)
{
	// if a is not divisible by b, return a/b + 1, else return a/b
	return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

/**
 * Get the last CUDA error and display it if required
 */
extern "C" void checkLastCudaError ()
{
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if ( error != hipSuccess )
	{
		printf("CUDA ERROR: %s\n", hipGetErrorString(error));
	}
}

/// entry point for the CUDA version of the BitmapAdd BitmapSubtract and BitmapMultiply plugin
extern "C" void bitmap_arithmetic_kernel_entry(int operation, unsigned short* p_deviceImage, int width, int height, float value)
{
    // allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));

	switch ( operation )
	{
    	case CUDA_BITMAP_ADD:
    		// execute the add
    		add_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, value);
    		break;
    	case CUDA_BITMAP_MULTIPLY:
    		// execute the multiply kernel
    		multiply_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, value);
    		break;
    	case CUDA_BITMAP_SUBTRACT:
    		// execute the add kernel with value negated
    		add_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, -value);
    		break;
        case CUDA_BITMAP_GAMMA:
            // execute the gamma kernel
            gamma_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, value);
            break;
        case CUDA_BITMAP_INVERT:
            // excute the bitmap invert kernel
            invert_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height);
            break;
        case CUDA_BITMAP_MATTE_COLORDIFF:
            matte_color_diff_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, value);
            break;
        case CUDA_BITMAP_MATTE_INVERT:
            matte_invert_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height);
            break;
    	default:
    		// unknown operation
    		;
	}

    // check if the kernel executed correctly
    checkLastCudaError();
    // Make sure this function blocks until the calculation is complete
    hipDeviceSynchronize();
}

extern "C" void bitmap_color_monochrome_kernel_entry(unsigned short* p_deviceImage, int width, int height, float redWeight, float greenWeight, float blueWeight)
{
	// allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));

	color_monochrome_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, redWeight, greenWeight, blueWeight);

    // check if the kernel executed correctly
    checkLastCudaError();
    hipDeviceSynchronize();

}

extern "C" void bitmap_threshold_kernel_entry(unsigned short* p_deviceImage, int width, int height, float redThreshold, float greenThreshold, float blueThreshold, float alphaThreshold)
{
    // allocate the blocks and threads
    dim3 threads_per_block(8, 8);
    dim3 blocks_per_grid( iDivUp(width, 8), iDivUp(height,8));

    threshold_kernel<<< blocks_per_grid, threads_per_block >>> ((ushort4*)p_deviceImage, width, height, redThreshold, greenThreshold, blueThreshold, alphaThreshold);

    // check if the kernel executed correctly
    checkLastCudaError();
    hipDeviceSynchronize();

}

extern "C" void copy_and_bind_texture_to_array( void** cudaArrayPointer, float* arrayData, int width, int height )
{
	// alocate a hipArray to store the transformation matrix
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray* cu_array;
	hipMallocArray( &cu_array, &channelDesc, width, height );
    hipMemcpyToArray( cu_array, 0, 0, arrayData, width*height*sizeof(float), hipMemcpyHostToDevice);

	// set texture parameters
    transformTexture.addressMode[0] = hipAddressModeClamp;
    transformTexture.addressMode[1] = hipAddressModeClamp;
    transformTexture.filterMode = hipFilterModePoint;
    transformTexture.normalized = false;

	// Bind the array to the texture
    hipBindTextureToArray( transformTexture, cu_array, channelDesc);

	*cudaArrayPointer = (void*)cu_array;
}

extern "C" void free_CUDA_array ( void* cudaArrayPointer )
{
	hipFreeArray((hipArray*)cudaArrayPointer);
}

extern "C" void apply_linear_transform_to_point_data ( float *device_points, float *T_matrix, int num_points )
{
	dim3 threads_per_block(64, 1);
    dim3 blocks_per_grid( iDivUp(num_points, 64), 1);

	linear_transform_kernel <<< blocks_per_grid, threads_per_block >>> ((float4*)device_points, num_points);

	// check if the kernel executed correctly
    checkLastCudaError();
    hipDeviceSynchronize();
}

extern "C" void allocate_device_memory ( void** device_pointer, int size_in_bytes )
{
	hipMalloc(device_pointer, size_in_bytes);
}

extern "C" void set_device_memory ( void* device_pointer, int value, int size_in_bytes )
{
	hipMemset(device_pointer, value, size_in_bytes);
}

extern "C" void resize_device_memory_block ( void** new_device_pointer, void* current_device_pointer, int new_size_in_bytes, int old_size_in_bytes, char clear )
{
	if ( new_size_in_bytes == old_size_in_bytes )
	{
		*new_device_pointer = current_device_pointer;
	}
	else
	{
		hipMalloc(new_device_pointer, new_size_in_bytes);
		hipDeviceSynchronize();

		if ( !clear )
		{
			if ( new_size_in_bytes < old_size_in_bytes )
			{
				hipMemcpy(*new_device_pointer, current_device_pointer, new_size_in_bytes, hipMemcpyDeviceToDevice);
			}
			else
			{
				hipMemset( *new_device_pointer, 0, new_size_in_bytes );
				hipDeviceSynchronize();
				hipMemcpy(*new_device_pointer, current_device_pointer, old_size_in_bytes, hipMemcpyDeviceToDevice);
			}
			hipDeviceSynchronize();
		}
		hipFree ( current_device_pointer );
	}

	if ( clear )
	{
		hipMemset( *new_device_pointer, 0, new_size_in_bytes );
		hipDeviceSynchronize();
	}
	else
	{
		hipDeviceSynchronize();
	}

}

extern "C" void copy_from_host_to_device ( void* device_pointer, const void* host_pointer, int size_in_bytes )
{
	hipMemcpy(device_pointer, host_pointer, size_in_bytes, hipMemcpyHostToDevice);
}

extern "C" void copy_from_host_to_device_64_to_32_convert ( void* device_pointer, const void* host_pointer, int size_in_bytes )
{
    #define NUM_THREADS 64
    int num_uints = size_in_bytes/sizeof(unsigned int);
    uint2* pdev_uint_64;

    allocate_device_memory((void**)&pdev_uint_64, size_in_bytes*2);
    copy_from_host_to_device((void*)pdev_uint_64, (const void*)host_pointer, size_in_bytes*2);

    dim3 threads_per_block(NUM_THREADS, 1);
    dim3 blocks_per_grid( iDivUp(num_uints, NUM_THREADS), 1);

    convert_uint_64_to_32_kernel <<< blocks_per_grid, threads_per_block >>> ( pdev_uint_64, (unsigned int*) device_pointer, num_uints);

    checkLastCudaError();

    hipDeviceSynchronize();
    free_device_memory ( pdev_uint_64 );
}

extern "C" void copy_from_device_to_host ( void* host_pointer, const void* device_pointer, int size_in_bytes )
{
	hipMemcpy(host_pointer, device_pointer, size_in_bytes, hipMemcpyDeviceToHost);
}

extern "C" void copy_from_device_to_host_32_to_64_convert ( void* host_pointer, const void* device_pointer, int size_in_bytes )
{
	#define NUM_THREADS 64
	int num_uints = size_in_bytes/sizeof(unsigned int);
	uint2* pdev_uint_64;

	allocate_device_memory((void**)&pdev_uint_64, size_in_bytes*2);

	dim3 threads_per_block(NUM_THREADS, 1);
	dim3 blocks_per_grid( iDivUp(num_uints, NUM_THREADS), 1);

	convert_uint_32_to_64_kernel <<< blocks_per_grid, threads_per_block >>> ( pdev_uint_64, (unsigned int*) device_pointer, num_uints);

	checkLastCudaError();

	copy_from_device_to_host(host_pointer, (const void*)pdev_uint_64, size_in_bytes*2);

	hipDeviceSynchronize();
	free_device_memory ( pdev_uint_64 );
}

extern "C" void copy_from_device_to_device ( void* device_dest_pointer, const void* device_source_pointer, int size_in_bytes )
{
    hipMemcpy(device_dest_pointer, device_source_pointer, size_in_bytes, hipMemcpyDeviceToDevice);
}
extern "C" void free_device_memory ( void* device_pointer )
{
	hipFree(device_pointer);
}

extern "C" void allocate_pinned_host_memory ( void** pointer_on_host, int size_in_bytes )
{
	hipHostMalloc(pointer_on_host, size_in_bytes);
}

extern "C" void free_pinned_host_memory ( void* pointer_on_host )
{
	hipHostFree(pointer_on_host);
}

extern "C" void transform_points_device_mesh ( float * pdev_output_points_and_selection, int num_points )
{
	dim3 threads_per_block(64, 1);
	dim3 blocks_per_grid( iDivUp(num_points, 64), 1);

	linear_transform_kernel <<< blocks_per_grid, threads_per_block >>> ((float4*)(pdev_output_points_and_selection), num_points);
	checkLastCudaError();
}

extern "C" void transform_points_synchronous ( double *InputPoints, double *PointSelection, double *OutputPoints, int num_points, timingInfo_t* tInfo )
{
	#define SETUP 0
	#define CONVERT_PRE 1
	#define TO_DEVICE 2
	#define EXECUTE 3
	#define TO_HOST 4
	#define CONVERT_POST 5
	#define CLEANUP 6

	// initialize the timing info structure
	initTimingInfo(tInfo, 7);

    setTimingInfoLabel(tInfo, SETUP, "SETUP");
	setTimingInfoLabel(tInfo, CONVERT_PRE, "CONVERT_PRE");
	setTimingInfoLabel(tInfo, TO_DEVICE, "TO_DEVICE");
	setTimingInfoLabel(tInfo, EXECUTE, "EXECUTE");
	setTimingInfoLabel(tInfo, TO_HOST, "TO_HOST");
	setTimingInfoLabel(tInfo, CONVERT_POST, "CONVERT_POST");
	setTimingInfoLabel(tInfo, CLEANUP, "CLEANUP");

	startTimingInfoTimer ( tInfo, SETUP );
    float *device_points;

	// allocate the memory on the device - 16 bytes per point
	allocate_device_memory((void**)&device_points, num_points*sizeof(float)*4);

	// allocate pinned host memory to allow for asynchronous operations
	float *host_points_single_p;
	allocate_pinned_host_memory ((void**)&host_points_single_p, num_points*sizeof(float)*4);

	dim3 threads_per_block(64, 1);
	dim3 blocks_per_grid( iDivUp(num_points, 64), 1);

	measureTimingInfoTimer( tInfo, SETUP );



    startTimingInfoTimer (tInfo, CONVERT_PRE);
	for (int point = 0; point < num_points; ++point)
	{
		int float_index = (point)*4;
		int double_index = (point)*3;
		host_points_single_p[float_index] = (float)InputPoints[double_index];
		host_points_single_p[float_index+1] = (float)InputPoints[double_index+1];
		host_points_single_p[float_index+2] = (float)InputPoints[double_index+2];
		host_points_single_p[float_index+3] = (float)PointSelection[point];
	}
	measureTimingInfoTimer (tInfo, CONVERT_PRE);


	startTimingInfoTimer (tInfo, TO_DEVICE);
	hipMemcpy(device_points, host_points_single_p, num_points*16, hipMemcpyHostToDevice);
	synchronize_threads();
	measureTimingInfoTimer (tInfo, TO_DEVICE);


	startTimingInfoTimer (tInfo, EXECUTE);
	linear_transform_kernel <<< blocks_per_grid, threads_per_block >>> ((float4*)(device_points), num_points);
	hipDeviceSynchronize();
	measureTimingInfoTimer ( tInfo, EXECUTE );

	startTimingInfoTimer ( tInfo, TO_HOST );
	hipMemcpy(host_points_single_p, device_points, num_points*16, hipMemcpyDeviceToHost);
	measureTimingInfoTimer ( tInfo, TO_HOST );

	startTimingInfoTimer ( tInfo, CONVERT_POST );
	for (int point = 0; point < num_points; ++point)
	{
		int float_index = (point)*4;
		int double_index = (point)*3;
		OutputPoints[double_index] = host_points_single_p[float_index];
		OutputPoints[double_index+1] = host_points_single_p[float_index+1];
		OutputPoints[double_index+2] = host_points_single_p[float_index+2];
	}
	measureTimingInfoTimer(tInfo, CONVERT_POST);

	startTimingInfoTimer ( tInfo, CLEANUP );
	free_device_memory(device_points);
	free_pinned_host_memory ( host_points_single_p );
	measureTimingInfoTimer ( tInfo, CLEANUP );
}

extern "C" void transform_points_asynchronous ( double *InputPoints, double *PointSelection, double *OutputPoints, int num_points, timingInfo_t* tInfo )
{
	#define SETUP 0
	#define STREAM_CREATE 1
	#define PHASE_1 2
	#define PHASE_2 3
	#define STREAM_DESTROY 4
	#define DEV_CLEANUP 5

	// initialize the timing info structure
	initTimingInfo(tInfo, 6);

    setTimingInfoLabel(tInfo, SETUP, "SETUP");
	setTimingInfoLabel(tInfo, STREAM_CREATE, "STREAM_CREATE");
	setTimingInfoLabel(tInfo, PHASE_1, "CONVERT_TO_DEVICE_EXECUTE");
	setTimingInfoLabel(tInfo, PHASE_2, "TO_HOST_CONVERT");
	setTimingInfoLabel(tInfo, STREAM_DESTROY, "STREAM_DESTROY");
	setTimingInfoLabel(tInfo, DEV_CLEANUP, "CLEANUP");



	startTimingInfoTimer ( tInfo, SETUP);
	// set the number of streams
	int nstreams = 4;

    float *device_points;
	// allocate the memory on the device - 16 bytes per point
	allocate_device_memory((void**)&device_points, num_points*sizeof(float)*4);

	// allocate pinned host memory to allow for asynchronous operations
	float *host_points_single_p;
	allocate_pinned_host_memory ((void**)&host_points_single_p, num_points*sizeof(float)*4);

	int points_per_stream = num_points/nstreams;

	dim3 threads_per_block(32, 1);
	dim3 blocks_per_grid( iDivUp(points_per_stream, 32), 1);
    measureTimingInfoTimer(tInfo, SETUP);

	startTimingInfoTimer ( tInfo, STREAM_CREATE);
	// allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t*) malloc(nstreams * sizeof(hipStream_t));
    for(int n = 0; n < nstreams; n++)
    	hipStreamCreate(&(streams[n]));
    measureTimingInfoTimer(tInfo, STREAM_CREATE);

    startTimingInfoTimer ( tInfo, PHASE_1);
	for ( int n = 0; n < nstreams; n++ )
	{
		// Convert a subset of the data to floats
		for (int point = n*points_per_stream; point < (n+1)*points_per_stream; ++point)
		{
			int float_index = (point)*4;
			int double_index = (point)*3;
			host_points_single_p[float_index] = (float)InputPoints[double_index];
			host_points_single_p[float_index+1] = (float)InputPoints[double_index+1];
			host_points_single_p[float_index+2] = (float)InputPoints[double_index+2];
			host_points_single_p[float_index+3] = (float)PointSelection[point];
		}

		// for each stream copy the data to the device and execute the kernel
		hipMemcpyAsync(device_points + n*points_per_stream*4, host_points_single_p + n*points_per_stream*4, points_per_stream*16, hipMemcpyHostToDevice, streams[n]);
		linear_transform_kernel <<< blocks_per_grid, threads_per_block, 0, streams[n] >>> ((float4*)(device_points + n*points_per_stream*4), points_per_stream);
	}
    measureTimingInfoTimer(tInfo, PHASE_1);

    startTimingInfoTimer ( tInfo, PHASE_2);
	// copy the data back from the device and convert
	for ( int n = 0; n < nstreams; n++ )
	{
		hipMemcpyAsync(host_points_single_p + n*points_per_stream*4, device_points + n*points_per_stream*4, points_per_stream*16, hipMemcpyDeviceToHost, streams[n]);
		// need to synchronize the streams so that the data is available to copy to the output points
		hipStreamSynchronize(streams[n]);
		for (int point = n*points_per_stream; point < (n+1)*points_per_stream; ++point)
		{
			int float_index = (point)*4;
			int double_index = (point)*3;
			OutputPoints[double_index] = host_points_single_p[float_index];
			OutputPoints[double_index+1] = host_points_single_p[float_index+1];
			OutputPoints[double_index+2] = host_points_single_p[float_index+2];
		}
	}
	measureTimingInfoTimer(tInfo, PHASE_2);

	startTimingInfoTimer ( tInfo, STREAM_DESTROY);
	// release resources
	for(int n = 0; n < nstreams; n++)
	{
    	hipStreamDestroy(streams[n]);
	}
	measureTimingInfoTimer(tInfo, STREAM_DESTROY);

	startTimingInfoTimer ( tInfo, DEV_CLEANUP);
	free_device_memory(device_points);
	free_pinned_host_memory ( host_points_single_p );
	measureTimingInfoTimer(tInfo, DEV_CLEANUP);
}

extern "C" void subdivide_edges_split_point_calculator ( const unsigned int* pdev_first_midpoint,
																												const unsigned char* pdev_has_midpoint,
                                                        float* pdev_points_and_selection,
                                                        unsigned int num_input_points,
                                                        unsigned int* pdev_edge_point_indices,
                                                        unsigned int* pdev_clockwise_edge_indices,
                                                        const float* pdev_edge_selection,
                                                        const unsigned int* pdev_companions,
                                                        const unsigned char* pdev_boundary_edges,
                                                        const unsigned int* pdev_edge_faces,
                                                        int num_split_points,
                                                        int num_edges)
{
    // allocate device memory for the edge_indices
    if ( num_edges > 0 & num_split_points > 0 & num_input_points > 0 )
    {
        int threads_x = 512 / num_split_points;

        dim3 threads_per_block(threads_x, num_split_points);
        dim3 blocks_per_grid( iDivUp(num_edges, threads_x), 1);
        subdivide_edges_split_point_kernel<<< blocks_per_grid, threads_per_block >>> ( pdev_first_midpoint,
                                                                                       pdev_has_midpoint,
                                                                                       (float4*)pdev_points_and_selection,
                                                                                       num_input_points,
                                                                                       pdev_edge_point_indices,
                                                                                       pdev_clockwise_edge_indices,
                                                                                       pdev_edge_selection,
                                                                                       pdev_companions,
                                                                                       pdev_boundary_edges,
                                                                                       pdev_edge_faces,
                                                                                       num_split_points,
                                                                                       num_edges);
        checkLastCudaError();
    }
}

extern "C" void subdivide_edges_update_indices_entry (unsigned int* pdev_input_edge_point_indices,
                                                      unsigned int* pdev_input_clockwise_edge_point_indices,
                                                      unsigned int num_host_edges,
                                                      unsigned int* pdev_output_edge_point_indices,
                                                      unsigned int* pdev_output_clockwise_edge_point_indices,
                                                      unsigned int* pdev_edge_index_map,
                                                      int num_edge_maps)
{
    int threads_x = 512;
    dim3 threads_per_block(threads_x, 1);
    dim3 blocks_per_grid( iDivUp(num_edge_maps, threads_x), 1);


    subdivide_edges_update_edge_indices_kernel<<< blocks_per_grid, threads_per_block >>>
                                              ( pdev_output_edge_point_indices,
                                                pdev_output_clockwise_edge_point_indices,
                                                pdev_input_edge_point_indices,
                                                pdev_input_clockwise_edge_point_indices,
                                                pdev_edge_index_map,
                                                num_edge_maps );

    checkLastCudaError();
}

extern "C" void subdivide_edges_update_loop_first_edges_entry (
                                                        unsigned int* pdev_ouput_loop_first_edges,
                                                        unsigned int num_loops,
                                                        unsigned int* pdev_edge_index_map,
                                                        int num_edge_maps
                                                            )
{
    int threads_x = 64;
    dim3 threads_per_block(threads_x, 1);
    dim3 blocks_per_grid( iDivUp(num_loops, threads_x), 1);

    subdivide_edges_update_loop_first_edges_kernel<<< blocks_per_grid, threads_per_block >>>
                                              ( pdev_ouput_loop_first_edges,
                                                num_loops,
                                                pdev_edge_index_map );

    checkLastCudaError();
}

extern "C" void subdivide_edges_split_edges_entry ( unsigned int* pdev_output_edge_point_indices,
                                                    unsigned int* pdev_output_clockwise_edge_point_indices,
                                                    unsigned int* pdev_input_clockwise_edge_point_indices,
                                                    const float* pdev_edge_selection,
                                                    unsigned int* pdev_edge_index_map,
                                                    unsigned int* pdev_first_midpoint,
                                                    unsigned char* pdev_has_midpoint,
                                                    int num_split_points,
                                                    unsigned int* pdev_companions,
                                                    unsigned char* pdev_boundary_edges,
                                                    const unsigned int num_edges,
                                                    const unsigned int* pdev_edge_faces
                                                    )
{
    int threads_x = 512 / num_split_points;
    dim3 threads_per_block(threads_x, num_split_points);
    dim3 blocks_per_grid( iDivUp(num_edges, threads_x), 1);

    subdivide_edges_split_edges_kernel<<< blocks_per_grid, threads_per_block >>>
                                                   (pdev_output_edge_point_indices,
                                                    pdev_output_clockwise_edge_point_indices,
                                                    pdev_input_clockwise_edge_point_indices,
                                                    pdev_edge_selection,
                                                    pdev_edge_index_map,
                                                    pdev_first_midpoint,
                                                    pdev_has_midpoint,
                                                    num_split_points,
                                                    pdev_companions,
                                                    pdev_boundary_edges,
                                                    num_edges,
                                                    pdev_edge_faces
                                                    );
    checkLastCudaError();
}

extern "C" void copy_2D_from_host_to_device_with_padding ( void* device_pointer, const void* host_pointer, int device_pitch, int host_pitch, int width_in_bytes, int rows )
{
    hipMemcpy2D(device_pointer, device_pitch, host_pointer, host_pitch, width_in_bytes, rows, hipMemcpyHostToDevice);
}

/**
 * Call thread synchronize to ensure consistency
 */
extern "C" void synchronize_threads ()
{
    hipDeviceSynchronize();
}

extern "C" void set_selection_value_entry ( float* points_and_selection, float selection_value, int num_points )
{
    int numThreads = 64;

    dim3 threads_per_block(numThreads, 1);
    dim3 blocks_per_grid( iDivUp(num_points, numThreads), 1);

    set_selection_value_kernel <<< blocks_per_grid, threads_per_block >>> ( (float4*)points_and_selection, selection_value, num_points );

    checkLastCudaError();

    hipDeviceSynchronize();
}

extern "C" void find_companion_kernel_entry ( unsigned char* pdev_boundary_edges,
											  unsigned int* pdev_adjacent_edge_indices,
											  const int num_edges,
											  const unsigned int* pdev_edge_point_indices,
											  const unsigned int* pdev_clockwise_edges_point_indices,
											  const unsigned int* pdev_first_edges,
											  const unsigned int* pdev_valences,
											  const unsigned int* pdev_point_edges )
{
	#define NUM_THREADS 64

	dim3 threads_per_block(NUM_THREADS, 1);
	dim3 blocks_per_grid( iDivUp(num_edges, NUM_THREADS), 1);

	// initialize the values for the boundary edges to true and the adjacent edge list to zeros
	hipMemset(pdev_boundary_edges, 1, num_edges*sizeof(unsigned char));
	hipMemset(pdev_adjacent_edge_indices, 0, num_edges*sizeof(unsigned int));
	hipDeviceSynchronize();

	find_companion_kernel <<< blocks_per_grid, threads_per_block >>> ( pdev_boundary_edges,
																	   pdev_adjacent_edge_indices,
																	   num_edges,
																	   pdev_edge_point_indices,
																	   pdev_clockwise_edges_point_indices,
																	   pdev_first_edges,
																	   pdev_valences,
																	   pdev_point_edges );

	checkLastCudaError();

	hipDeviceSynchronize();


}

extern "C" int create_vertex_valence_lookup_kernel_entry (
																unsigned int* pdev_valence,
																const unsigned int* pdev_edge_point_indices,
																int num_edges
																)
{
	int host_valence_size = 0;
	int* pdev_valence_size;
	allocate_device_memory((void**)&pdev_valence_size, sizeof(int));

	// intialize the valence data on the device - set to zero
	hipMemset((void*)pdev_valence, 0, num_edges*2*sizeof(unsigned int));

	// the following runs serially

	dim3 threads_per_block(1, 1);
	dim3 blocks_per_grid(1, 1);

	hipDeviceSynchronize();
	create_vertex_valence_lookup_kernel<<< blocks_per_grid, threads_per_block >>> ( pdev_valence, pdev_edge_point_indices, pdev_valence_size, num_edges );
	hipDeviceSynchronize();
	checkLastCudaError();

	copy_from_device_to_host((void*)&host_valence_size, (const void*)pdev_valence_size, sizeof(int));


	free_device_memory((void*)pdev_valence_size);

	return host_valence_size;
}

extern "C" void calculate_first_edge_entry ( unsigned int* pdev_first_edge, const unsigned int* pdev_valences, int num_points )
{
	// Serial execution
	int numThreads = 1;
	dim3 threads_per_block(numThreads, 1);
	dim3 blocks_per_grid( 1, 1);

	calculate_first_edges_kernel<<< blocks_per_grid, threads_per_block >>> ( pdev_first_edge, pdev_valences, num_points );
	checkLastCudaError();

}

extern "C" void calculate_point_edges_entry (
											unsigned int* pdev_point_edges,
											unsigned int* pdev_found_edges,
											const unsigned int* pdev_edge_point_indices,
											const unsigned int* pdev_first_edges,
											int num_edges,
											int num_points )
{
	// initialize the point edges
	hipMemset((void*)pdev_point_edges, 0, num_edges*sizeof(unsigned int));
	hipMemset((void*)pdev_found_edges, 0, num_points*sizeof(unsigned int));
	// runs serially
	dim3 threads_per_block(1, 1);
	dim3 blocks_per_grid(1, 1);
	hipDeviceSynchronize();
	calculate_point_edges_kernel<<<blocks_per_grid, threads_per_block>>> (pdev_point_edges, pdev_found_edges, pdev_edge_point_indices, pdev_first_edges, num_edges);
	checkLastCudaError();
}

extern "C" void edge_index_calculator_entry (
													unsigned int* pdev_first_midpoint,
													unsigned char* pdev_has_midpoint,
													unsigned int* pdev_index_map,
													unsigned int* pdev_edge_faces,
													const unsigned int* pdev_face_first_loops,
													const unsigned int* pdev_face_loop_counts,
													const unsigned int* pdev_loop_first_edges,
													const unsigned int* pdev_clockwise_edges,
													const float* pdev_edge_selection,
													const unsigned int* pdev_companions,
													const unsigned char* pdev_boundary_edges,
													int split_point_count,
													int num_faces,
													int first_new_point_index,
													unsigned int* new_point_count,
													unsigned int* new_edge_count
													)
{
	dim3 threads_per_block(64, 1);
	dim3 blocks_per_grid( iDivUp(num_faces, 64), 1);

	// Get the face for each edge
	face_per_edge_kernel <<< blocks_per_grid, threads_per_block >>> (
										pdev_face_first_loops,
										num_faces,
										pdev_face_loop_counts,
										pdev_loop_first_edges,
										pdev_clockwise_edges,
										pdev_edge_faces);
	checkLastCudaError();
	hipDeviceSynchronize();
	
	// Allocate an array to store the per-face edge and new split point count
	unsigned int* face_edge_counts;
	allocate_device_memory((void**)&face_edge_counts, num_faces * sizeof(unsigned int));
	unsigned int* face_split_point_counts;
	allocate_device_memory((void**)&face_split_point_counts, num_faces * sizeof(unsigned int));

	// count edges and new split points per face
	subdivide_edges_count_components_per_face <<< blocks_per_grid, threads_per_block >>> (
										pdev_face_first_loops,
										num_faces,
										pdev_face_loop_counts,
										pdev_loop_first_edges,
										pdev_clockwise_edges,
										pdev_edge_selection,
										pdev_companions,
										pdev_boundary_edges,
										split_point_count,
										pdev_edge_faces,
										face_edge_counts,
										face_split_point_counts
										 );
	hipDeviceSynchronize();
	
	// Replace face_edge_counts with its cumulative sum
	unsigned int* temp_e;
	allocate_device_memory((void**)&temp_e, num_faces * sizeof(unsigned int));
	unsigned int* temp_p;
	allocate_device_memory((void**)&temp_p, num_faces * sizeof(unsigned int));
	for(int step_size = 1; step_size < num_faces; step_size *= 2)
	{
		unsigned int* temp2_e = face_edge_counts;
		face_edge_counts = temp_e;
		temp_e = temp2_e;
		cumulative_sum_step<<< blocks_per_grid, threads_per_block >>>(temp_e, face_edge_counts, num_faces, step_size);
		checkLastCudaError();
		unsigned int* temp2_p = face_split_point_counts;
		face_split_point_counts = temp_p;
		temp_p = temp2_p;
		cumulative_sum_step<<< blocks_per_grid, threads_per_block >>>(temp_p, face_split_point_counts, num_faces, step_size);
		checkLastCudaError();
		hipDeviceSynchronize();
	}
	
	// Calculate the new indices for the split points and the edges
	subdivide_edges_edge_index_calculator_kernel <<< blocks_per_grid, threads_per_block >>> (
										pdev_first_midpoint,
										pdev_has_midpoint,
										pdev_index_map,
										pdev_face_first_loops,
										num_faces,
										pdev_face_loop_counts,
										pdev_loop_first_edges,
										pdev_clockwise_edges,
										pdev_edge_selection,
										pdev_companions,
										pdev_boundary_edges,
										pdev_edge_faces,
										face_edge_counts,
										face_split_point_counts,
										split_point_count,
										first_new_point_index
										 );
	checkLastCudaError();

	hipDeviceSynchronize();
	
	copy_from_device_to_host((void*) new_point_count, (void*) &(face_split_point_counts[num_faces - 1]), sizeof(unsigned int));
	copy_from_device_to_host((void*) new_edge_count, (void*) &(face_edge_counts[num_faces - 1]), sizeof(unsigned int));
			
	free_device_memory(temp_e);
	free_device_memory(temp_p);
	free_device_memory(face_edge_counts);
	free_device_memory(face_split_point_counts);
}

/**
 * entry point for setting up a polygrid on the device
 */
extern "C" void create_grid_structure_kernel_entry (
												unsigned int* pdev_face_first_loops,
												unsigned int* pdev_face_loop_count,
												unsigned int* pdev_loop_first_edge,
												unsigned int* pdev_edge_point,
												unsigned int* pdev_clockwise_edge,
												unsigned int rows,
												unsigned int columns)
{
	dim3 threads_per_block(8, 8);

	dim3 blocks_per_grid(iDivUp(rows, 8), iDivUp(columns, 8));

	create_grid_structure_kernel<<< blocks_per_grid, threads_per_block >>> (
												pdev_face_first_loops,
												pdev_face_loop_count,
												pdev_loop_first_edge,
												(uint4*) pdev_edge_point,
												(uint4*) pdev_clockwise_edge,
												rows,
												columns );
	checkLastCudaError();

}

/**
 * Entry point for calculating polyGrid point positions
 */
extern "C" void calculate_grid_points_kernel_entry (
												float* pdev_point_and_selection,
												float* phost_x,
												float* phost_y,
												unsigned int rows,
												unsigned int columns
												)
{
	dim3 threads_per_block(8, 8);

	dim3 blocks_per_grid(iDivUp(rows, 8), iDivUp(columns, 8));

	float3 x;
	x.x = phost_x[0];
	x.y = phost_x[1];
	x.z = phost_x[2];
	float3 y;
	y.x = phost_y[0];
	y.y = phost_y[1];
	y.z = phost_y[2];

	calculate_grid_points_kernel <<< blocks_per_grid, threads_per_block >>> (
															(float4*) pdev_point_and_selection,
															x,
															y,
															rows,
															columns
															);
	checkLastCudaError();

}

